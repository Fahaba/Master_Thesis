#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <crt/math_functions.h>
#include<winsock2.h>
#pragma comment(lib,"ws2_32.lib") //Winsock Library

#define THRESHHOLD 25000
#define CUDACORES 2560
#define MAX_THREADS_PER_BLOCK 1024
#define PI2 6.2832
#define channelNumber 4
#define sampleSize 4

typedef hipComplex cplx;
__device__ __forceinline__ cplx cuexpf(cplx z)
{
    cplx res;
    float t = expf(z.x);
    sincosf(z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

__device__ void _dft(cplx* buf, cplx* out, int N, dim3 blockIdx, dim3 blockDimx, dim3 gridDim, dim3 threadIdx)
{
    double PI = acosf(-1);

    int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
    int side = gridDim.x * blockIdx.y * blockDim.x;
    int sideIndex = blockIdx.x * blockDim.x + threadNumInBlock;
    int globalIndex = side + sideIndex; //global thread number;

    for (int j = 0; j < N; j++)
    {
        int index_j = side + j;
        cplx exp = make_hipComplex(cos(((2 * PI) / N) * sideIndex * j), -1 * sin(((2 * PI) / N) * sideIndex * j));
        cplx res = hipCaddf(out[globalIndex], hipCmulf(buf[index_j], exp));
        out[globalIndex] = res;
    }
}

__device__ __forceinline__ void _dft2(cplx* buf, cplx* out, int n, int channelNb, int N)
{
    double PI =  acosf(-1);

    for (int channel = 0; channel < channelNumber; channel++)
    {
        for (int i = 0; i < n; i++)
        {
            for (int j = 0; j < n; j++)
            {
                int index_i = channel * n + i;
                int index_j = channel * n + j;
                cplx exp = make_hipComplex(cos(((2 * PI) / N) * i * j), -1 * sin(((2 * PI) / N) * i * j));
                out[index_i] = hipCaddf(out[index_i], hipCmulf(buf[index_j], exp));
            }
        }
    }
}

__device__ __forceinline__ void _fft(cplx* buf, cplx* out, int n, int step)
{
    double PI = acosf(-1);
    if (step < n) {
        _fft(out, buf, n, step * 2);
        _fft(out + step, buf + step, n, step * 2);

        for (int j = 0; j < n; j += 2 * step) {

            cplx exp = hipCmulf(cuexpf(make_hipComplex(PI * j / n, -1)), out[j + step]);
            
            buf[j / 2] = hipCaddf(out[j], exp);
            buf[(j + n) / 2] = hipCsubf(out[j], exp);
        }
    }
}

void printbuf(cplx* buf_orig, cplx* buf, int size, int threshholdCount, int channelNb, int** x) {
    
    int n = 0;
    FILE* f;

    fopen_s(&f, "..\\out\\dftplots2.m", "a");

    // make this generic with channelnumber
    
        for (n = 0; n < size; ++n)
        {
            for (int channel = 0; channel < channelNumber; channel++)
            {
                fprintf(f, "%f ", hipCrealf(buf_orig[channel * size + n]));
                float dftMagnitude = sqrtf(hipCrealf(buf[channel * size + n]) * hipCrealf(buf[channel * size + n]) + 
                    hipCimagf(buf[channel * size + n]) * hipCimagf(buf[channel * size + n]));
                fprintf(f, "%f ", dftMagnitude);
            }
            fprintf(f, "%u\n", threshholdCount * size + n);
        }
        printf("Wrote %u magnitudes at %u channel(s)\n", size, channelNb);

    fclose(f);
}

__device__ __forceinline__ void fft(int* buf, cplx* out, int n)
{
    cplx* buf2 = (cplx*)malloc(sizeof(cplx) * n);
    for (int j = 1; j < n; j++) {
        printf("buf[%i]=%i\n", j, buf[j]);
        cplx one = make_hipComplex(buf[j], 0);
        buf2[j] = one;
    }
    for (int j = 0; j < n; j++)
    {
        printf("cOut[%i]=real(%f) im(%f) \n", j, hipCrealf(buf2[j]), hipCimagf(buf2[j]));
    }
    _fft(buf2, out, n, 1);
    
}

hipError_t cudaFourierTransform(cplx* input, cplx* complexOut, int channelNb, int size);

__global__ void addKernel(cplx* x, cplx* out, int channelNummber, long size)
{
    //unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    _dft(x, out, size, blockIdx, blockDim, gridDim, threadIdx);
}

//FFT(x) {
//    n = length(x);
//    if (n == 1) return x;
//    m = n / 2;
//    X = (x_{ 2j })_ { j = 0 }^{m - 1};
//    Y = (x_{ 2j + 1 })_ { j = 0 }^{m - 1};
//    X = FFT(X);
//    Y = FFT(Y);
//    U = (X_{ k mod m })_ { k = 0 }^{n - 1};
//    V = (g^{ -k }Y_{ k mod m })_ { k = 0 }^{n - 1};
//    return U + V;
//}

int convertToInt(char* buf, int bufsize)
{
    /*int shift = 8;
    int res = buf[0] & 0xFF;
    for (int i = 0; i < bufsize; ++i)
    {
        res |= (buf[i] & i < (bufsize - 1) ? 0xFF : buf[i]) << shift;
        shift += 8;
    }

    return res;*/

    int shift = 8;
    int ret = buf[0] & 0xFF;
    for (int i = 1; i < bufsize; ++i)
    {
        char one = buf[1];
        ret |= (buf[i] & i < (bufsize - 1) ? 0xFF : buf[i]) << shift;
        shift += 8;
    }
    return ret;
}

int sendBuf(SOCKET s, cplx* complx, int threshhold, int channelNb)
{
    float* mag = (float*)malloc(sizeof(float) * threshhold * channelNumber);
    int msgsize = 1024;
    char* msg = (char*)malloc(sizeof(char) * msgsize);
    msg[0] = '[';
    int length = 1;
    for (int c = 0; c < channelNumber; c++)
    {
        for (int samp = 0; samp < threshhold; samp++)
        {
            float ftMagnitude = sqrtf(
                  hipCrealf(complx[c * threshhold + samp]) 
                * hipCrealf(complx[c * threshhold + samp]) 
                + hipCimagf(complx[c * threshhold + samp]) 
                * hipCimagf(complx[c * threshhold + samp]));
            mag[c * threshhold + samp] = ftMagnitude;
            
            int len = snprintf(NULL, 0, "%f", ftMagnitude) +1;

            if (length + len > msgsize)
            {
                msgsize += 100;
                msg = (char*)realloc(msg, msgsize * sizeof(char));
            }

            if (c == channelNumber - 1 && samp == threshhold-1)
                sprintf(msg + length, "%f]", ftMagnitude);
            else
                sprintf(msg + length, "%f,", ftMagnitude);
            
            length += len;
        }
    }
    // build message
    char datalength[1024];
    sprintf(datalength, "%u|", strlen(msg));
    send(s, datalength, strlen(datalength), 0);
    send(s, msg, strlen(msg), 0);

    return 0;
}

bool initSocket(SOCKET& s, char* host, int port)
{
    struct sockaddr_in server;
    WSADATA wsa;
    printf("\nInitialising Winsock...");
    if (WSAStartup(MAKEWORD(2, 2), &wsa) != 0)
    {
        printf("Failed. Error Code : %d", WSAGetLastError());
        return false;
    }

    printf("Initialized.");

    if ((s = socket(AF_INET, SOCK_STREAM, 0)) == INVALID_SOCKET)
    {
        printf("Could not create socket : %d", WSAGetLastError());
    }

    printf("Socket created.\n");

    server.sin_addr.s_addr = inet_addr(host);
    server.sin_family = AF_INET;
    server.sin_port = htons(port);

    //Connect to remote server
    if (connect(s, (struct sockaddr*)&server, sizeof(server)) < 0)
    {
        printf("Could not create connection : %d", WSAGetLastError());
        puts("connect error");
        return false;
    }

    puts("Connected");
    return true;
}

int main()
{
    SOCKET s_vis;
    SOCKET s_rcv;

    if (!initSocket(s_vis, "127.0.0.1", 1338))
        return -1;
    if (!initSocket(s_rcv, "127.0.0.1", 1337))
        return -1;

    int** sampleData = (int**)malloc(sizeof(int*) * channelNumber);
    for (int i = 0; i < channelNumber; i++)
        sampleData[i] = (int*)malloc(sizeof(int) * THRESHHOLD);

    int arr_pos = 0;
    long start = 0;
    int rcvSize = (channelNumber * sampleSize + channelNumber - 1 + 1) * THRESHHOLD;
    char* rcvBuf = (char*)malloc(sizeof(char) * rcvSize);
    char* startptr, *endptr;
    while (true)
    {
        if (arr_pos >= THRESHHOLD)
        {
            cplx* complexOut = (cplx*)malloc(sizeof(cplx) * channelNumber * THRESHHOLD);
            cplx* helper = (cplx*)malloc(sizeof(cplx) * channelNumber * THRESHHOLD);

            for (int i = 0; i < channelNumber; i++)
            {
                for (int j = 0; j < THRESHHOLD; j++)
                {
                    int sample = sampleData[i][j];
                    helper[i * THRESHHOLD + j] = make_hipComplex(sampleData[i][j], 0);
                }
            }
            // cuda
            cudaFourierTransform(helper, complexOut, channelNumber, THRESHHOLD);
            sendBuf(s_vis, complexOut, THRESHHOLD, channelNumber);

            start = arr_pos;
            arr_pos = 0;

            // realloc sampleData array
            free(sampleData);
            sampleData = (int**)malloc(sizeof(int*) * channelNumber);
            for (int i = 0; i < channelNumber; i++)
                sampleData[i] = (int*)malloc(sizeof(int) * THRESHHOLD);
        }
        

        int bytes = 0;

        while (bytes < rcvSize)
        {
            int recvBytes = recv(s_rcv, rcvBuf + bytes, rcvSize - bytes, 0);
            bytes += recvBytes;
        }

        char delim[] = "\n";
        
        unsigned count = 0;
        startptr = endptr = (char*)rcvBuf;

        //                                                  2               4   = 8 +      " "        "\n"
        char* channelStr = (char*)malloc(sizeof(char) * (channelNumber * sampleSize + (sampleSize-1) + 1));
        while ((endptr = strchr(startptr, '\n')))
        {
            sprintf(channelStr, "%.*s", (int)(endptr - startptr + 1), startptr);
            char* perChannel = strtok(channelStr, " ");
            for (int c = 0; c < channelNumber; c++)
            {
                int channelData = (int)strtol(perChannel, NULL, 16);
                perChannel = strtok(NULL, " ");
                sampleData[c][arr_pos] = channelData;
            }
            startptr = endptr + 1;
            arr_pos++;
        }
        
    }
    free(rcvBuf);
    return 0;
}

hipError_t cudaFourierTransform(cplx* x, cplx* complexOut, int channelNb, int size)
{
    hipError_t cudaStatus;

    cplx* dev_c;
    cplx* dev_out;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    // Allocate GPU buffers
    cudaStatus = hipMalloc((void**)&dev_out, sizeof(cplx) * channelNumber * size);
    cudaStatus = hipMalloc((void**)&dev_c, sizeof(cplx) * channelNumber * size);
    cudaStatus = hipMemcpy(dev_c, x, sizeof(cplx) * channelNumber * size, hipMemcpyHostToDevice);

    int maxThreadsPerBlock = 1024;
    int threadCluster = (int)ceilf((float)size / maxThreadsPerBlock);
    int actualThreadsPerBlock = size;

    if (threadCluster)
        actualThreadsPerBlock = (int)ceilf((float)size / threadCluster);
    
    dim3 block_dim(actualThreadsPerBlock, 1, 1);
    dim3 grid_dim(threadCluster, channelNumber, 1);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel <<<grid_dim, block_dim >>> (dev_c, dev_out, channelNumber, size);
    
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(complexOut, dev_out, sizeof(cplx) * size * channelNumber, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    printf("DFT done with sitze %u\n", size);

Error:
    hipFree(dev_out);
    hipFree(dev_c);

    return cudaStatus;
}
